#include <iostream>
#include <fstream>
#include <limits>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

// Function to calculate the Frobenius norm of a matrix
double frobeniusNormCPU(const std::vector<std::vector<int>>& matrix) {
    double sum = 0.0;
    for (int i = 0; i < matrix.size(); i++) {
        for (int j = 0; j < matrix[i].size(); j++) {
            int element = matrix[i][j];
            sum += element * element;
        }
    }
    return std::sqrt(sum);
}

// Function to calculate the similarity score using Cosine similarity
double calculateSimilarityScoreCPU(const std::vector<std::vector<int>>& graph1, const std::vector<std::vector<int>>& graph2) {
    // Check if the graphs have the same size
    if (graph1.size() != graph2.size() || graph1[0].size() != graph2[0].size()) {
        std::cerr << "Error: Graphs must have the same size." << std::endl;
        return -1;
    }

    // Calculate the Frobenius norms of the adjacency matrices
    double normGraph1 = frobeniusNormCPU(graph1);
    double normGraph2 = frobeniusNormCPU(graph2);

    // Calculate the similarity score
    double similarityScore = 0.0;

    for (size_t i = 0; i < graph1.size(); ++i) {
        for (size_t j = 0; j < graph1[0].size(); ++j) {
            similarityScore += (graph1[i][j] * graph2[i][j]) / (normGraph1 * normGraph2);
        }
    }

    return similarityScore;
}

// Function to print a graph (adjacency matrix)
void printMatrix(const std::vector<std::vector<int>>& graph, const std::string& label) {
    std::cout << "Graph " << label << ":" << std::endl;
    for (int i = 0; i < graph.size(); i++) {
        for (int j = 0; j < graph[i].size(); j++) {
            std::cout << graph[i][j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}


std::vector<std::vector<int>> generateAdjacencyMatrix(const std::string& filename) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        return {};
    }

    // Read the number of vertices from the first line
    unsigned long numVertices;
    file >> numVertices;

    // Initialize an adjacency matrix of the appropriate size with all zeros
    std::vector<std::vector<int>> adjacencyMatrix(numVertices, std::vector<int>(numVertices, 0));
    unsigned long edgepernode[numVertices];

    // Skip the next numVertices lines
    unsigned long totalEdges, numEdges;
    for (unsigned long i = 0; i < numVertices; ++i) {
        file >> totalEdges >> numEdges;
        edgepernode[i] = numEdges;
    }

    // Skip the next blank line
    file.ignore(std::numeric_limits<std::streamsize>::max(), '\n');

    // Skip the next line containing a single number
    unsigned long skip;
    file >> skip;

    // Skip the next blank line
    file.ignore(std::numeric_limits<std::streamsize>::max(), '\n');

    // Read the total number of edges from the next line
    file >> totalEdges;

    // Read the edges and fill in the adjacency matrix
    unsigned long dest;
    int weight;
    for (unsigned long i = 0; i < numVertices; ++i) {
        for(int j = 0; j < edgepernode[i]; ++j){
            // printf("Reading edge %d of vertex %lu\n", j, i);
            file >> dest >> weight;
            adjacencyMatrix[i][dest] = weight;
        }
    }

    file.close();
    return adjacencyMatrix;
}

std::vector<std::vector<int>> oppositeMatrix(const std::vector<std::vector<int>>& matrix) {
    int rows = matrix.size();
    int cols = matrix[0].size();
    std::vector<std::vector<int>> opposite(cols, std::vector<int>(rows));

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            opposite[i][j] = !matrix[i][j];
        }
    }

    return opposite;
}

std::vector<std::vector<int>> halfsameMatrix(const std::vector<std::vector<int>>& matrix) {
    int rows = matrix.size();
    int cols = matrix[0].size();
    std::vector<std::vector<int>> halfsame(rows, std::vector<int>(cols));

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols / 2; j++) {
            halfsame[i][j] = matrix[i][j];
        }
        for (int j = cols / 2; j < cols; j++) {
            halfsame[i][j] = !matrix[i][j];  // Generate random values
        }
    }
    
    return halfsame;
}

std::vector<std::vector<int>> generateRandomAdjacencyMatrix(int vertices, int weight_range) {
    std::vector<std::vector<int>> matrix(vertices, std::vector<int>(vertices));

    for (int i = 0; i < vertices; ++i) {
        for (int j = i + 1; j < vertices; ++j) {
            matrix[i][j] = rand() % weight_range;
            matrix[j][i] = matrix[i][j];  // For undirected graph
        }
    }

    return matrix;
}


// CUDA kernel to calculate the Frobenius norm of a matrix
__global__ void frobeniusNormKernel(int* matrix, int rows, int cols, double* result) {
    // TODO: Implement the kernel to calculate the Frobenius norm of a matrix
    return;
}

// Function to calculate the Frobenius norm of a matrix using CUDA
double frobeniusNormGPU(const std::vector<std::vector<int>>& matrix) {
    // TODO: Allocate memory on the GPU, copy the matrix to the GPU, launch the kernel,
    // copy the result back to the CPU, and free the GPU memory
    return -1.0;
}


// CUDA kernel to calculate the similarity score using
__global__ void calculateSimilarityScoreKernel(int* graph1, int* graph2, int rows, int cols, double normGraph1, double normGraph2, double* result) {
    // TODO: Implement the kernel to calculate the similarity score
    return;
}

// Function to calculate the similarity score using  with CUDA
double calculateSimilarityScoreGPU(const std::vector<std::vector<int>>& graph1, const std::vector<std::vector<int>>& graph2) {
    // TODO: Allocate memory on the GPU, copy the graphs to the GPU, launch the kernel,
    // copy the result back to the CPU, and free the GPU memory
    return -1.0;
}


int main(int argc, char* argv[]) {

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " [num_vertices_test] [input_file_1(optional)] [input_file_2(optional)]" << std::endl;
        return 1;
    }

    int TEST_VERTICES = std::stoi(argv[1]);

    // Test Case 1: Identical matrices, should result in score 1
    std::vector<std::vector<int>> identicalMatrix1 = generateRandomAdjacencyMatrix(TEST_VERTICES, 10);
    std::vector<std::vector<int>> identicalMatrix2 = identicalMatrix1;

    // // Test Case 2: Completely different matrices, should result in score 0
    std::vector<std::vector<int>> oppositeMatrix1 = generateRandomAdjacencyMatrix(TEST_VERTICES, 10);
    std::vector<std::vector<int>> oppositeMatrix2 = oppositeMatrix(oppositeMatrix1);

    // Test Case 3: Half of the elements are the same, should result in score 0.5
    std::vector<std::vector<int>> halfsameMatrix1 = generateRandomAdjacencyMatrix(TEST_VERTICES, 2);
    std::vector<std::vector<int>> halfsameMatrix2 = halfsameMatrix(halfsameMatrix1);

    double similarityScoreCPU = -1.0;
    double similarityScoreGPU = -1.0;

    // Test #1
    printf("Test 1: Identical matrices (expected score 1)\n");
    similarityScoreCPU = calculateSimilarityScoreCPU(identicalMatrix1, identicalMatrix2);
    similarityScoreGPU = calculateSimilarityScoreGPU(identicalMatrix1, identicalMatrix2);
    printf("CPU Similarity score: %.2f\n", similarityScoreCPU);
    printf("GPU Similarity score: %.2f\n", similarityScoreGPU);
    if (similarityScoreCPU > 0.99 && abs(similarityScoreCPU - similarityScoreGPU) < 0.01) {
        std::cout << "Test Case 1: Passed" << std::endl << std::endl;
    } else {
        printMatrix(identicalMatrix1, "1");
        printMatrix(identicalMatrix2, "2");
        std::cout << "Test Case 1: Failed" << std::endl << std::endl;
        exit(1);
    }

    // Test #2
    printf("Test 2: Opposite matrices (expected score 0)\n");
    similarityScoreCPU = calculateSimilarityScoreCPU(oppositeMatrix1, oppositeMatrix2);
    similarityScoreGPU = calculateSimilarityScoreGPU(oppositeMatrix1, oppositeMatrix2);
    printf("CPU Similarity score: %.2f\n", similarityScoreCPU);
    printf("GPU Similarity score: %.2f\n", similarityScoreGPU);
    if (similarityScoreCPU == 0.0 && abs(similarityScoreCPU - similarityScoreGPU) < 0.01) {
        std::cout << "Test Case 2: Passed" << std::endl << std::endl;
    } else {
        printMatrix(oppositeMatrix1, "1");
        printMatrix(oppositeMatrix2, "2");
        std::cout << "Test Case 2: Failed" << std::endl << std::endl; 
        exit(1);
    }

    // Test #3
    printf("Test 3: Half same matrices (expected score 0.5)\n");
    similarityScoreCPU = calculateSimilarityScoreCPU(halfsameMatrix1, halfsameMatrix2);
    similarityScoreGPU = calculateSimilarityScoreGPU(halfsameMatrix1, halfsameMatrix2);
    printf("CPU Similarity score: %.2f\n", similarityScoreCPU);
    printf("GPU Similarity score: %.2f\n", similarityScoreGPU);
    if (similarityScoreCPU > 0.49 && similarityScoreCPU < 0.51 && abs(similarityScoreCPU - similarityScoreGPU) < 0.01) {
        std::cout << "Test Case 3: Passed" << std::endl << std::endl;
    } else {
        printMatrix(halfsameMatrix1, "1");
        printMatrix(halfsameMatrix2, "2");
        std::cout << "Test Case 3: Failed" << std::endl << std::endl;
        exit(1);
    }

    if(argc > 2) {
        std::string inputFile1 = argv[2];
        std::string inputFile2 = argv[3];
        std::string inputDir   = "./inputGen/";

        printf("Test 4: Custom graphs\n");
    
        // Example adjacency matrices for two graphs
        std::vector<std::vector<int>> adjacencyMatrix1 = generateAdjacencyMatrix(inputDir+inputFile1);
        std::vector<std::vector<int>> adjacencyMatrix2 = generateAdjacencyMatrix(inputDir+inputFile2);

        // Calculate the similarity matrix
        similarityScoreCPU = calculateSimilarityScoreCPU(adjacencyMatrix1, adjacencyMatrix2);
        similarityScoreGPU = calculateSimilarityScoreGPU(adjacencyMatrix1, adjacencyMatrix2);

        if (abs(similarityScoreCPU - similarityScoreGPU) < 0.01) {
            std::cout << "Custom Graph Test: Passed" << std::endl << std::endl;
        } else {
            printMatrix(adjacencyMatrix1, "1");
            printMatrix(adjacencyMatrix2, "2");
            std::cout << "Custom Graph Test: Failed" << std::endl << std::endl;
            exit(1);
        }
    }

    return 0;
}
